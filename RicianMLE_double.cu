/* Rician MLE diffusion and kurtosis tensor estimator by Viljami Sairanen (2016)
Based on algorithm in:
"Liu, Jia, Dario Gasbarra, and Juha Railavo. 
"Fast Estimation of Diffusion Tensors under
Rician noise by the EM algorithm."
Journal of neuroscience methods 257 (2016) : 147 - 158" */

// to convert between single and double precision use following changes:
//  double <-> double
// sqrt( <-> sqrt(
// fabs( <-> fabs(
//  exp( <-> exp(
//  log( <-> log(


#include <hip/hip_runtime.h>
#include <math.h>

__device__ size_t calculateGlobalIndex() {
	// Which block are we?
	size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
	// Which THREAD are we within the block?
	size_t const localthreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
	// How big is each block?
	size_t const threadsPerBlock = blockDim.x*blockDim.y;
	// Which THREAD are we overall?
	return localthreadIdx + globalBlockIndex*threadsPerBlock;
}
__device__  double getBesseli0(double x) {
	double ax, ans, y;
	ax = fabs(x);
	if (ax < 3.75) {
		y = x / 3.75;
		y *= y;
		ans = 1.0 + y*(3.5156229 + y*(3.0899424 + y*(1.2067492 +
			y*(0.2659732 + y*(0.360768e-1 + y*0.45813e-2)))));
		ans *= exp(-ax); // scale by exp(-abs(real(x))); see matlab help for besseli
	}
	else {
		y = 3.75 / ax;
		ans = (1.0 / sqrt(ax)) * // scale by exp(-abs(real(x))); see matlab help for besseli
			(0.39894228 + y * (0.1328592e-1
				+ y * (0.225319e-2 + y * (-0.157565e-2 + y * (0.916281e-2
					+ y * (-0.2057706e-1 + y * (0.2635537e-1 + y * (-0.1647633e-1
						+ y * (0.392377e-2)))))))));
	}
	return ans;
}
__device__  double getBesseli1(double x) {
	double ax, ans, y;
	ax = fabs(x);
	if (ax < 3.75) {
		y = x / 3.75;
		y *= y;
		ans = ax * (0.5 + y *(0.87890594 + y *(0.51498869 + y *(0.15084934
			+ y * (0.2658733e-1 + y * (0.301532e-2 + y * 0.32411e-3))))));
		ans *= exp(-ax); // scale by exp(-abs(real(x))); see matlab help for besseli
	}
	else {
		y = 3.75 / ax;
		ans = 0.2282967e-1 + y * (-0.2895312e-1 + y * (0.1787654e-1
			- y * 0.420059e-2));
		ans = 0.39894228 + y * (-0.3988024e-1 + y * (-0.362018e-2
			+ y * (0.163801e-2 + y * (-0.1031555e-1 + y * ans))));
		ans *= 1.0 / sqrt(ax); // scale by exp(-abs(real(x))); see matlab help for besseli
	}
	return x < 0.0 ? -ans : ans;
}
__device__ double getMax(
	double *arr,
	const unsigned int length,
	size_t const THREAD) {
	double ans;
	ans = arr[THREAD * length];
	for (int i = 1; i < length; i++) {
		if (arr[THREAD * length + i] > ans) {
			ans = arr[THREAD * length + i];
		}
	}
	return ans;
}
__device__  void LUdecomposition(double *a, int n, int *indx, double *vv, size_t const THREAD) {
	int i, imax, j, k;
	double big, dum, sum, temp;

	for (i = 0; i<n; i++) {
		big = 0.0;
		for (j = 0; j<n; j++) {
			temp = fabs(a[THREAD * n * n+ i*n + j]);
			if (temp >= big) {
				big = temp;
			}
		}
		if (big == 0.0) { // Singular matrix can't compute
			big = 1.0e-20;
		}
		vv[THREAD * n + i] = 1.0 / big;
	}
	for (j = 0; j<n; j++) {
		for (i = 0; i<j; i++) {
			sum = a[THREAD * n * n+ i*n + j];
			for (k = 0; k<i; k++) {
				sum -= a[THREAD * n * n+ i*n + k] * a[THREAD * n * n+ k*n + j];
			}
			a[THREAD * n * n+ i*n + j] = sum;
		}
		big = 0.0;
		for (i = j; i<n; i++) {
			sum = a[THREAD * n * n+ i*n + j];
			for (k = 0; k<j; k++) {
				sum -= a[THREAD * n * n+ i*n + k] * a[THREAD * n * n+ k*n + j];
			}
			a[THREAD * n * n+ i*n + j] = sum;
			dum = vv[THREAD * n+ i] * fabs(sum);
			if (dum >= big) {
				big = dum;
				imax = i;
			}
		}
		if (j != imax) {
			for (k = 0; k<n; k++) {
				dum = a[THREAD * n * n+ imax*n + k];
				a[imax*n + k] = a[THREAD * n * n+ j*n + k];
				a[THREAD * n * n+ j*n + k] = dum;
			}
			vv[THREAD * n+ imax] = vv[THREAD * n+ j];
		}
		indx[THREAD * n+ j] = imax;
		if (a[THREAD * n * n+ j*n + j] == 0.0) {
			a[THREAD * n * n+ j*n + j] = 1.0e-20;
		}
		if (j != n) {
			dum = 1.0 / a[THREAD * n * n+ j*n + j];
			for (i = j + 1; i<n; i++) {
				a[THREAD * n * n+ i*n + j] *= dum;
			}
		}
	}
}
__device__  void LUsubstitutions(double *a, int n, int *indx, double *b, size_t const THREAD) {
	int i, ii = 0, ip, j;
	double sum;
	for (i = 0; i<n; i++) {
		ip = indx[(THREAD * n) + i];
		sum = b[(THREAD * n) + ip];
		b[(THREAD * n) + ip] = b[(THREAD * n) + i];
		if (ii != 0) {
			for (j = ii - 1; j<i; j++) {
				sum -= a[(THREAD * n * n) + (i * n) + j] * b[(THREAD * n) + j];
			}
		}
		else if (sum != 0) {
			ii = i + 1;
		}
		b[(THREAD * n) + i] = sum;
	}
	for (i = n - 1; i >= 0; i--) {
		sum = b[(THREAD * n) + i];
		for (j = i + 1; j<n; j++) {
			sum -= a[(THREAD * n * n) + (i * n) + j] * b[(THREAD * n) + j];
		}
		b[(THREAD * n) + i] = sum / a[(THREAD * n * n) + (i * n) + i];
	}
}
__device__ void CholeskyDecomposition(double *a, int n, double *p, size_t const THREAD) {
	int i, j, k;
	double sum;
	for (i = 0; i < n; i++) {
		for (j = i; j < n; j++) {
			sum = a[(THREAD * n * n) + (i*n) + j];
			for (k = i-1; k >= 0; k--) {
				sum -= a[(THREAD * n * n) + (i*n) + k]
					* a[(THREAD * n * n) + (j*n) + k];
			}
			if (i == j) {
				if (sum <= 0.0) {
					sum = 1.0e-20; // Cholesky decomposition failed
				}
				p[THREAD*n + i] = sqrt(sum);
			}
			else {
				a[(THREAD*n*n) + (j*n) + i] = sum / p[THREAD*n + i];
			}
		}
	}
}
__device__ void CholeskyBacksubstitution(double *a, int n, double *p, double *b, double *x, size_t const THREAD) {
	int i, k;
	double sum;
	for (i = 0; i < n; i++) { // Solve Ly=b, storing y in x
		sum = b[THREAD*n + i];
		for (k = i-1; k >= 0; k--) {
			sum -= a[(THREAD*n*n) + (i*n) + k] * x[THREAD*n + k];
		}
		x[THREAD*n + i] = sum / p[THREAD*n + i];
	}
	for (i = n; i >= 0; i--) { // Solve L^(T)x=y
		sum = x[THREAD*n + i];
		for (k = i+1; k < n; k++) {
			sum -= a[(THREAD*n*n) + (k*n) + i] * x[THREAD*n + k];
		}
		x[THREAD*n + i] = sum / p[THREAD*n + i];
	}
}
__device__ void calculateExpZTheta(
	double *expZTheta, 
	double *theta, 
	double *Z,
	const unsigned int nParams, 
	const unsigned int nDWIs,
	size_t const THREAD) {

	for (int i = 0; i < nDWIs; i++) {
		expZTheta[THREAD * nDWIs + i] = 0.0;
		for (int j = 0; j < nParams; j++) {
			expZTheta[THREAD * nDWIs + i] +=
				Z[j * nDWIs + i] * theta[THREAD * nParams + j];
		}
		expZTheta[THREAD * nDWIs + i] = exp(expZTheta[THREAD * nDWIs + i]);
	}

}
__device__ void calculateAB_1(
	double *a,
	double *b,
	double *Y,
	double *expZTheta,
	double *sumYSQ,
	const unsigned int nDWIs,
	size_t const THREAD) {

	a[THREAD] = sumYSQ[THREAD];
	for (int i = 0; i < nDWIs; i++) {
		a[THREAD] += expZTheta[THREAD * nDWIs + i] * expZTheta[THREAD * nDWIs + i];
		b[THREAD * nDWIs + i] = Y[THREAD * nDWIs + i] * expZTheta[THREAD * nDWIs + i];
	}

}
__device__ void calculateAB_2(
	double *a,
	double *b,
	double *Y,
	double *Z,
	double *theta,
	double *SigmaSQ,
	double *expZTheta,
	double *twotau,
	const unsigned int nDWIs,
	const unsigned int nParams,
	size_t const THREAD) {
	// Now indexing for i ranges [0, nDWIs-1] and j ranges [1, nParams] since first nParams is the theta(1)
	a[THREAD] = 0.0;
	for (int i = 0; i < nDWIs; i++) {
		expZTheta[THREAD * nDWIs + i] = 0.0;
		for (int j = 1; j < nParams; j++) {
			expZTheta[THREAD * nDWIs + i] +=
				Z[j * nDWIs + i] * theta[THREAD * nParams + j];
		}
		expZTheta[THREAD * nDWIs + i] = exp(expZTheta[THREAD * nDWIs + i]);
		a[THREAD] += expZTheta[THREAD * nDWIs + i] * expZTheta[THREAD * nDWIs + i];
		b[THREAD * nDWIs + i] = Y[THREAD * nDWIs + i] * expZTheta[THREAD * nDWIs + i];
		twotau[THREAD * nDWIs + i] = b[THREAD * nDWIs + i] * exp(theta[THREAD * nParams+0]) / SigmaSQ[THREAD];
	}
	a[THREAD] = log(a[THREAD]);
}
__device__ void calculateEN(
	double *EN,
	double *twotau,
	const unsigned int nDWIs,
	bool *anyEN,
	size_t const THREAD) {
	anyEN[THREAD] = false;
	for (int i = 0; i < nDWIs; i++) {
		EN[THREAD * nDWIs + i] = 0.5 * twotau[THREAD * nDWIs + i] *
			getBesseli1(twotau[THREAD * nDWIs + i]) /
			getBesseli0(twotau[THREAD * nDWIs + i]);
		if (EN[THREAD * nDWIs + i] > 0.0) {
			anyEN[THREAD] = true;
		}
	}
}
__device__ void calculateZTheta(
	double *c,
	double *ZTheta,
	double *theta,
	double *SigmaSQ,
	double *Z,
	const unsigned int nDWIs,
	const unsigned int nParams,
	size_t const THREAD) {
	// Now indexing for i ranges [0, nDWIs-1] and j ranges [1, nParams] since first nParams is the theta(1)
	c[THREAD] = 2.0 * theta[THREAD * nParams+0] - 
		log(2.0 * SigmaSQ[THREAD]);
	for (int i = 0; i < nDWIs; i++) {
		ZTheta[THREAD * nDWIs + i] = 0.0;
		for (int j = 1; j < nParams; j++) {
			ZTheta[THREAD * nDWIs + i] +=
				Z[j * nDWIs + i] * theta[THREAD * nParams + j];
		}
		ZTheta[THREAD * nDWIs + i] *= 2.0;
		ZTheta[THREAD * nDWIs + i] += c[THREAD];
	}
}
__device__ void calculateLoglikelihood(
	double *loglikelihood,
	double *expo,
	double *ZTheta,
	double *scaling,
	double *expScaling,
	double *EN,
	const unsigned int nDWIs,
	size_t const THREAD) {
	loglikelihood[THREAD] = 0.0;
	for (int i = 0; i < nDWIs; i++) {
		expo[THREAD * nDWIs + i] = exp(ZTheta[THREAD * nDWIs + i] - scaling[THREAD]);
		loglikelihood[THREAD] +=
			EN[THREAD * nDWIs + i] * ZTheta[THREAD * nDWIs + i]
			- expo[THREAD * nDWIs + i] * expScaling[THREAD];
	}
}
__device__ void initializeInformationMatrices(
	double *fisherInformation,
	double *fisherInformation_sym,
	const unsigned int nDeltaParams,
	size_t const THREAD) {
	for (int i = 0; i < nDeltaParams*nDeltaParams; i++) {
		fisherInformation[THREAD * nDeltaParams*nDeltaParams + i] = 0.0;
		fisherInformation_sym[THREAD * nDeltaParams*nDeltaParams + i] = 0.0;
	}
}
__device__ void iterateSigmaSQ(
	double *SigmaSQ,
	double *SigmaSQ0,
	double *tmpdouble,
	double *a,
	double *b,
	double *twotau,
	unsigned int *nIterSigmaSQ,
	unsigned int iterLimitSigmaSQ,
	const double toleranceSigmaSQ,
	const unsigned int nDWIs,
	bool *continueSigmaSQIteration,
	size_t const THREAD) {
	// Should be ok
	continueSigmaSQIteration[THREAD] = true;
	nIterSigmaSQ[THREAD] = 0;
	
	while (continueSigmaSQIteration[THREAD]) {		
		(nIterSigmaSQ[THREAD])++;
		SigmaSQ0[THREAD] = SigmaSQ[THREAD];
		tmpdouble[THREAD] = 0.0;
		for (int i = 0; i < nDWIs; i++) {
			twotau[THREAD * nDWIs + i] = b[THREAD * nDWIs + i] / SigmaSQ[THREAD];
			tmpdouble[THREAD] += twotau[THREAD * nDWIs + i] * 
				getBesseli1(twotau[THREAD * nDWIs + i]) / 
				getBesseli0(twotau[THREAD * nDWIs + i]);
		}
		SigmaSQ[THREAD] = 0.5 * a[THREAD] / ((double)(nDWIs) + tmpdouble[THREAD]);
			
		continueSigmaSQIteration[THREAD] =
			((nIterSigmaSQ[THREAD] < iterLimitSigmaSQ)
				&&
			(fabs(SigmaSQ[THREAD] - SigmaSQ0[THREAD]) > toleranceSigmaSQ));
	}

}
__device__ void iterateS0(
	double *theta,
	double *theta1_old,
	double *SigmaSQ,
	double *a,
	double *b,
	double *twotau,
	unsigned int *nIterS0,
	unsigned int iterLimitS0,
	const double toleranceS0,
	const unsigned int nDWIs,
	const unsigned int nParams,
	bool *continueS0Iteration,
	size_t const THREAD) {
	
	continueS0Iteration[THREAD] = true;
	nIterS0[THREAD] = 0;

	while (continueS0Iteration[THREAD]) {
		nIterS0[THREAD]++;
		// Get initial theta(1) parameter
		theta1_old[THREAD] = theta[THREAD * nParams+0];
		// Calculate new theta(1) parameter
		theta[THREAD * nParams+0] = 0.0;
		for (int i = 0; i < nDWIs; i++) {
			theta[THREAD * nParams+0] += (b[THREAD * nDWIs + i] *
				getBesseli1(twotau[THREAD * nDWIs + i]) /
				getBesseli0(twotau[THREAD * nDWIs + i]));
		}
		theta[THREAD * nParams+0] = log(theta[THREAD * nParams+0]) -a[THREAD];
		// Update twotau for the next iteration step
		for (int i = 0; i < nDWIs; i++) {
			twotau[THREAD * nDWIs + i] = b[THREAD * nDWIs + i] *
				exp(theta[THREAD * nParams+0]) / SigmaSQ[THREAD];
		}
		// Test to end while loop

		continueS0Iteration[THREAD] =
			((nIterS0[THREAD] < iterLimitS0)
				&&
				(fabs((theta[THREAD * nParams + 0] - theta1_old[THREAD]) / theta1_old[THREAD])));
	}
}
__device__ void calculateFisherInformation( 
	double *fisherInformation,
	double *fisherInformation_sym,
	double *Z,
	double *score,
	double *DeltaTheta,
	double *expo,
	double *EN,
	double *expScaling,
	const unsigned int nDWIs,
	const unsigned int nParams,
	const unsigned int nDeltaParams,
	size_t const THREAD) {

	for (int j = 1; j < nParams; j++) {
		score[THREAD * nDeltaParams + j - 1] = 0.0;
		for (int i = 0; i < nDWIs; i++) {
			score[THREAD * nDeltaParams + j - 1] +=
				2.0 * Z[j * nDWIs + i] * (EN[THREAD * nDWIs + i] -
					expo[THREAD * nDWIs + i] * expScaling[THREAD]);
			for (int k = 1; k < nParams; k++) { // range of j and k are [1 to nParams]
				fisherInformation[THREAD * nDeltaParams*nDeltaParams + (j - 1)*nDeltaParams + (k - 1)] +=
					4.0 * Z[j * nDWIs + i] * Z[k * nDWIs + i] * expo[THREAD * nDWIs + i];
				// Symmetrize Fisher Information
				fisherInformation_sym[THREAD * nDeltaParams*nDeltaParams + (j - 1)*nDeltaParams + (k - 1)] =
					(fisherInformation[THREAD * nDeltaParams*nDeltaParams + (j - 1)*nDeltaParams + (k - 1)] +
					 fisherInformation[THREAD * nDeltaParams*nDeltaParams + (j - 1)*nDeltaParams + (k - 1)]) *
					0.5 * expScaling[THREAD];
			}
		}
		DeltaTheta[THREAD * nDeltaParams + j - 1] = score[THREAD *nDeltaParams + j - 1];
	}
	// Make copy of symmetric Fisher information matrix
	for (int i = 0; i < nDeltaParams*nDeltaParams; i++) {
		fisherInformation[THREAD * nDeltaParams * nDeltaParams + i] = fisherInformation_sym[THREAD * nDeltaParams * nDeltaParams + i];
	}
}
__device__ void iterateLoglikelihood(
	int *indx,
	double *score,
	double *vv,
	double *DeltaTheta,
	double *Z,
	double *expo,
	double *theta,
	double *loglikelihood,
	double *loglikelihood_old,
	double *new_theta,
	double *regulatorLambda,
	double *fisherInformation,
	double *fisherInformation_sym,
	double *ZTheta,
	double *c,
	double *scaling,
	double *expScaling,
	double *EN,
	const unsigned int nDWIs,
	const unsigned int nParams,
	const unsigned int nDeltaParams,
	const double regulatorLambda0,
	const double regulatorRescaling,
	unsigned int *nIterLoglikelihood,
	const unsigned int iterLimitLoglikelihood,
	const double toleranceLoglikelihood,
	bool *continueLoglikelihoodIteration,
	size_t const THREAD) {

	nIterLoglikelihood[THREAD] = 0;
	continueLoglikelihoodIteration[THREAD] = true;
	regulatorLambda[THREAD] = regulatorLambda0;
	while (continueLoglikelihoodIteration[THREAD]) {
		nIterLoglikelihood[THREAD]++;
		//loglikelihood_old[THREAD] = loglikelihood[THREAD]; // loglikelihood_old is not supposed to be updated in this loop
		// Initialize DeltaTheta for LUdecomposition & substitutions
		// because X = I\score calculated using LUsubstitutions actually
		// replaces values in score and we don't want to loose that information
		// so we have to save score into DeltaTheta variable
		for (int j = 1; j < nParams; j++) {
			DeltaTheta[THREAD * nDeltaParams + j - 1] = score[THREAD *nDeltaParams + j - 1];
		}
		// Regularize Fisher information matrix with lambda
		for (int i = 0; i < nDeltaParams; i++) {
			fisherInformation[THREAD * nDeltaParams*nDeltaParams + i*nDeltaParams + i] =
				fisherInformation_sym[THREAD * nDeltaParams*nDeltaParams + i*nDeltaParams + i]
				+ regulatorLambda[THREAD];
		}
		// Update regulatorLambda
		regulatorLambda[THREAD] *= regulatorRescaling;
		
		//LUdecomposition(fisherInformation, nDeltaParams, indx, vv, THREAD);
		//LUsubstitutions(fisherInformation, nDeltaParams, indx, DeltaTheta, THREAD);
		CholeskyDecomposition(fisherInformation, nDeltaParams, vv, THREAD);
		CholeskyBacksubstitution(fisherInformation, nDeltaParams, vv, score, DeltaTheta, THREAD);
		//goto THE_END_LOGLIKELIHOOD;
		// Calculate new theta(2:end)
		for (int i = 1; i < nParams; i++) {
			new_theta[THREAD * nDeltaParams + i - 1] =
				theta[THREAD * nParams + i] 
				+ DeltaTheta[THREAD * nDeltaParams + i - 1];
		}
		// Calculate ZTheta based on new_theta
		for (int i = 0; i < nDWIs; i++) {
			ZTheta[THREAD * nDWIs + i] = 0.0;
			for (int j = 1; j < nParams; j++) {
				ZTheta[THREAD * nDWIs + i] +=
					Z[j* nDWIs + i] * new_theta[THREAD * nDeltaParams + j - 1];
			}
			ZTheta[THREAD * nDWIs + i] *= 2.0;
			ZTheta[THREAD * nDWIs + i] += c[THREAD]; // c is based on theta(1) and sigmasq that are constant in this loop
		}
		scaling[THREAD] = getMax(ZTheta, nDWIs, THREAD);
		expScaling[THREAD] = exp(scaling[THREAD]);

		// Calculate new loglikelihood
		// calculateLoglikelihood updates loglikelihood and expo variables
		calculateLoglikelihood(loglikelihood, expo, ZTheta, scaling, expScaling, EN, nDWIs, THREAD);

		// Check if new loglikelihood is NaN, if so more regulation is needed
		// (f != f) is true only if f is NaN (IEEE standard)
		if (loglikelihood[THREAD] != loglikelihood[THREAD]) {
			// loglikelihood is NaN, check only iterations
			continueLoglikelihoodIteration[THREAD] = (nIterLoglikelihood[THREAD] < iterLimitLoglikelihood);
		}
		else {
			continueLoglikelihoodIteration[THREAD] =
				((loglikelihood[THREAD] < loglikelihood_old[THREAD])
					&&
					(nIterLoglikelihood[THREAD] < iterLimitLoglikelihood));
		}
	}
	//THE_END_LOGLIKELIHOOD:
}
__device__ void iterateTheta(
	int *indx,
	double *vv,
	double *theta,
	double *ZTheta,
	double *c,
	double *fisherInformation,
	double *fisherInformation_sym,
	double *score,
	double *Z,
	double *EN,
	double *scaling,
	double *expScaling,
	double *expo,
	double *DeltaTheta,
	double *DeltaThetaScore,
	double *new_theta,
	double *loglikelihood,
	double *loglikelihood_old,
	double *regulatorLambda,
	const double regulatorLambda0,
	const double regulatorRescaling,
	const unsigned int nDWIs,
	const unsigned int nParams,
	const unsigned int nDeltaParams,
	unsigned int *nIterTheta,
	unsigned int *nIterLoglikelihood,
	const unsigned int iterLimitTheta,
	const unsigned int iterLimitLoglikelihood,
	const double toleranceTheta,
	const double toleranceLoglikelihood,
	bool *continueThetaIteration,
	bool *continueLoglikelihoodIteration,
	size_t const THREAD) {
	// Now indexing for i ranges [0, nDWIs-1] and j ranges [1, nParams] since first nParams is the theta(1)
	continueThetaIteration[THREAD] = true;
	nIterTheta[THREAD] = 0;
	loglikelihood_old[THREAD] = loglikelihood[THREAD];
	while (continueThetaIteration[THREAD]) {
		nIterTheta[THREAD]++;
		calculateFisherInformation(fisherInformation, fisherInformation_sym, Z, score, DeltaTheta, expo, EN, expScaling, nDWIs, nParams, nDeltaParams, THREAD);
		
		// Optimize loglikelihood
		iterateLoglikelihood(indx, score, vv, DeltaTheta, Z, expo, theta, loglikelihood, loglikelihood_old, new_theta, regulatorLambda, fisherInformation, fisherInformation_sym, ZTheta, c, scaling, expScaling, EN, nDWIs, nParams, nDeltaParams, regulatorLambda0, regulatorRescaling, nIterLoglikelihood, iterLimitLoglikelihood, toleranceLoglikelihood, continueLoglikelihoodIteration, THREAD);
		//goto THE_END_THETA;
		DeltaThetaScore[THREAD] = 0.0;
		for (int i = 0; i < nDeltaParams; i++) {
			DeltaThetaScore[THREAD] += DeltaTheta[THREAD * nDeltaParams + i]
				* score[THREAD * nDeltaParams + i];
		}
		
		// Check if new loglikelihood is NaN, if not 
		// update theta(2:end) and loglikelihood_old
		if (loglikelihood[THREAD] != loglikelihood[THREAD]) {
		// NaN, don't update variables
			continueThetaIteration[THREAD] = (nIterTheta[THREAD] < iterLimitTheta);
		} else {
			for (int i = 1; i < nParams; i++) {
				theta[THREAD * nParams + i] = new_theta[THREAD * nDeltaParams + i - 1];
			}
			loglikelihood_old[THREAD] = loglikelihood[THREAD];
		
		continueThetaIteration[THREAD] =
			(((DeltaThetaScore[THREAD] > toleranceTheta)
				||
				((loglikelihood[THREAD] - loglikelihood_old[THREAD]) > toleranceLoglikelihood))
				&&
				(nIterTheta[THREAD] < iterLimitTheta));
		}
	}
	//THE_END_THETA:
}

__device__ void calculateNorms(
	double *norm1,
	double *norm2,
	double *theta,
	double *theta_old,
	const unsigned int nParams,
	size_t const THREAD) {
	
	norm1[THREAD] = 0.0;
	norm2[THREAD] = 0.0;
	for (int i = 0; i < nParams; i++) {
		norm1[THREAD] += theta_old[THREAD * nParams + i] * theta_old[THREAD * nParams + i];
		norm2[THREAD] += (theta[THREAD * nParams + i] - theta_old[THREAD * nParams + i])*
			(theta[THREAD * nParams + i] - theta_old[THREAD * nParams + i]);
	}
	norm1[THREAD] = sqrt(norm1[THREAD]);
	norm2[THREAD] = sqrt(norm2[THREAD]);
}

__global__ void RicianMLE(
	double *theta,
	double *SigmaSQ,
	double *Z,
	double *fisherInformation,
	double *fisherInformation_sym,
	double *score,
	double *DeltaTheta,
	double *new_theta,
	double *vv,
	int *indx,
	double *theta_old,
	double *Y,
	double *expZTheta,
	double *ZTheta,
	double *twotau,
	double *expo,
	double *EN,
	double *b,
	double *a,
	double *c,
	double *sumYSQ,
	double *theta1_old,
	double *SigmaSQ0,
	double *SigmaSQ_old,
	double *tmpdouble,
	double *scaling,
	double *expScaling,
	double *loglikelihood,
	double *loglikelihood_old,
	double *regulatorLambda,
	double *DeltaThetaScore,
	double *norm1,
	double *norm2,
	unsigned int *nIterSigmaSQ,
	unsigned int *nIterVoxel,
	unsigned int *nIterS0,
	unsigned int *nIterTheta,
	unsigned int *nIterLoglikelihood,
	bool *continueSigmaSQIteration,
	bool *continueVoxelIteration,
	bool *continueS0Iteration,
	bool *continueThetaIteration,
	bool *continueLoglikelihoodIteration,
	bool *anyEN,
	const double toleranceSigmaSQ,
	const double toleranceS0,
	const double toleranceTheta,
	const double toleranceLoglikelihood,
	const unsigned int iterLimitSigmaSQ,
	const unsigned int iterLimitVoxel,
	const unsigned int iterLimitS0,
	const unsigned int iterLimitTheta,
	const unsigned int iterLimitLoglikelihood,
	const double regulatorLambda0,
	const double regulatorRescaling,
	const unsigned int nDWIs,
	const unsigned int nParams,
	const unsigned int nDeltaParams,
	const unsigned int nVoxels) {
	
	// Initial, work out which THREAD i.e. voxel we are computing
	size_t const THREAD = calculateGlobalIndex();
	if (THREAD >= nVoxels) {
		return;
	}
	
	// First, optimize Rician loglikelihood w.r.t. SigmaSQ
	calculateExpZTheta( expZTheta, theta, Z, nParams, nDWIs, THREAD);
	calculateAB_1(a, b, Y, expZTheta, sumYSQ, nDWIs, THREAD);
	iterateSigmaSQ(SigmaSQ, SigmaSQ0, tmpdouble, a, b, twotau, nIterSigmaSQ, iterLimitSigmaSQ, toleranceSigmaSQ, nDWIs, continueSigmaSQIteration, THREAD);
	
	// Start voxel-wise optimization
	continueVoxelIteration[THREAD] = true;
	while (continueVoxelIteration[THREAD]) {
		nIterVoxel[THREAD]++;
		// Save initial theta and SigmaSQ to be used later to test if voxel optimization continues
		SigmaSQ_old[THREAD] = SigmaSQ[THREAD];
		for (int i = 0; i < nParams; i++) {
			theta_old[THREAD * nParams + i] = theta[THREAD * nParams + i];
		}
		// Second, optimize w.r.t. S0 i.e. theta(1) with fixed theta(2:end) and SigmaSQ				
		// calcuateAB_2 updates a,b, expZTheta, and twotau variables
		calculateAB_2(a, b, Y, Z, theta, SigmaSQ, expZTheta, twotau, nDWIs, nParams, THREAD);
		// iterateS0 updates theta(1) and twotau variables
		iterateS0(theta, theta1_old, SigmaSQ, a, b, twotau, nIterS0, iterLimitS0, toleranceS0, nDWIs, nParams, continueS0Iteration, THREAD);
		
		// Third, optimize w.r.t. theta(2:end) with fixed theta(1) and SigmaSQ
		// calculateEN updates conditional expectation EN and checks if any(EN > 0) 
		calculateEN(EN, twotau, nDWIs, anyEN, THREAD);
		
		if (anyEN[THREAD]) {
			// There is information to estimate tensor(s)
			// calculateZTheta updates c and ZTheta variables
			calculateZTheta(c, ZTheta, theta, SigmaSQ, Z, nDWIs, nParams, THREAD);
			scaling[THREAD] = getMax(ZTheta, nDWIs, THREAD);
			expScaling[THREAD] = exp(scaling[THREAD]);			
			// calculateLoglikelihood updates loglikelihood and expo variables
			calculateLoglikelihood(loglikelihood, expo, ZTheta, scaling, expScaling, EN, nDWIs, THREAD);			
			initializeInformationMatrices(fisherInformation, fisherInformation_sym, nDeltaParams, THREAD);			
			iterateTheta(indx, vv, theta, ZTheta, c, fisherInformation, fisherInformation_sym, score, Z, EN, scaling, expScaling, expo, DeltaTheta, DeltaThetaScore, new_theta, loglikelihood, loglikelihood_old, regulatorLambda, regulatorLambda0, regulatorRescaling, nDWIs, nParams, nDeltaParams, nIterTheta, nIterLoglikelihood, iterLimitTheta, iterLimitLoglikelihood, toleranceTheta, toleranceLoglikelihood, continueThetaIteration, continueLoglikelihoodIteration, THREAD);					
			//goto THE_END;
		}
		else {
			// There is no information for estimations
			// Set theta(2:end) and information to zero
			for (int i = 1; i < nParams; i++) {
				theta[THREAD * nParams + i] = 0.0;
			}
			initializeInformationMatrices(fisherInformation, fisherInformation_sym, nDeltaParams, THREAD);
		}
		
		// Last, optimize w.r.t. SigmaSQ with fixed theta
		calculateExpZTheta(expZTheta, theta, Z, nParams, nDWIs, THREAD);
		calculateAB_1(a, b, Y, expZTheta, sumYSQ, nDWIs, THREAD);
		iterateSigmaSQ(SigmaSQ, SigmaSQ0, tmpdouble, a, b, twotau, nIterSigmaSQ, iterLimitSigmaSQ, toleranceSigmaSQ, nDWIs, continueSigmaSQIteration, THREAD);
		
		calculateNorms(norm1, norm2, theta, theta_old, nParams, THREAD);

		continueVoxelIteration[THREAD] =
			(((fabs((SigmaSQ[THREAD] - SigmaSQ_old[THREAD]) / SigmaSQ_old[THREAD]) > toleranceSigmaSQ)
				||
				((norm2[THREAD] / norm1[THREAD]) > toleranceTheta))
				&&
				(nIterVoxel[THREAD] < iterLimitVoxel));
	}
	//THE_END:
}